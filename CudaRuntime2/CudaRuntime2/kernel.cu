#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include <locale.h>

#include <stdlib.h>

#include <hip/hip_runtime.h>

#include ""

#include<iostream>

#include<cmath>
using namespace std;


__global__ void calc(double* a, int n) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	double val = a[idx];

	if (idx < n) {

		a[idx] = sqrt(1 - val * val);

	}

}

int main() {


	double* a_h; 

	double* a_d;

	const int numSteps = 10000000; 

	a_h = (double*)malloc(sizeof(double) * numSteps);

	hipMalloc((void**)&a_d, sizeof(double) * numSteps); 

	int blockSize = 192;

	int blocks = numSteps / blockSize + (numSteps % blockSize == 0 ? 0 : 1);

	double left = 0.0;

	double right = 1.0;

	double step = (right - left) / numSteps;

	int i = 0;

	for (double x = left + 0.5 * step; x < right; x += step) {

		a_h[i] = x;

		i++;

	}

	hipMemcpy(a_d, a_h, sizeof(double) * numSteps, hipMemcpyHostToDevice);


	calc << < blocks, blockSize >> > (a_d, numSteps);

	hipMemcpy(a_h, a_d, sizeof(double) * numSteps, hipMemcpyDeviceToHost); 

	double sum = 0.0;

	for (int i = 0; i < numSteps; i++) {

		sum += a_h[i];

	}

	

	cout<<"PI="<< 4 * sum / numSteps;



	system("pause");

	free(a_h);

	hipFree(a_d);


}