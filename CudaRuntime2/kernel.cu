﻿#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include <stdio.h>
using namespace std;

int main() {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	cout<<"Device name"<<" "<<deviceProp.name<<endl;
	cout<<"Total global memory :  MB"<<" "<<deviceProp.totalGlobalMem / 1024 / 1024<<endl;
	cout<<"Shared memory per block : "<<deviceProp.sharedMemPerBlock<<endl;
	cout<<"Registers per block :"<<deviceProp.regsPerBlock<<endl;
	cout<<"Warp size :"<< deviceProp.warpSize<<endl;
	cout<<"Memory pitch :"<< deviceProp.memPitch<<endl;
	cout<<"Max threads per block :"<<deviceProp.maxThreadsPerBlock<<endl;
	printf("Max threads dimensions : x = %d, y = %d, z = % d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
	printf("Max grid size: x = %d, y = %d, z = %d\n",
		deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
		deviceProp.maxGridSize[2]);
	cout<<"Clock rate: "<<deviceProp.clockRate<<endl;
	cout<<"Total constant memory:"<<deviceProp.totalConstMem<<endl;
	printf("Compute capability: %d.%d\n",
		deviceProp.major, deviceProp.minor);
	cout<<"Texture alignment: "<<deviceProp.textureAlignment<<endl;
	cout<<"Device overlap:"<<" "<<deviceProp.deviceOverlap<<endl;
	cout<<"Multiprocessor count: "<<deviceProp.multiProcessorCount<<endl;
		cout<<"Kernel execution timeout enabled:"<<deviceProp.kernelExecTimeoutEnabled ? "true" :"false";

}