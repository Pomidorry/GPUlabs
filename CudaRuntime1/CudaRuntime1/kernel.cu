#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include <stdio.h>
using namespace std;
__global__ void ab(int a, int b, int* c) {
	*c = a + b;
}
__global__ void Enter()
{
	printf("Enter 2 numbers\n");
}
int main() {
	int a, b;
	Enter << <1, 1 >> > ();
	cin >> a >> b;
	int c;
	int* c1;
	hipMalloc((void**)&c1, sizeof(int));
	ab << <1, 1 >> > (a, b, c1);
	hipMemcpy(&c, c1, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
	cout << c;
	return 0;
}